#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "gputime.h"
#include <hip/hip_runtime.h>

#define N 256
#define ARRAY_SIZE 4096

using namespace std;
//Naive smooth kernel without shared memory
__global__ void smooth_naive(float * v, float * v_new) {
	int myIdx = threadIdx.x * gridDim.x + blockIdx.x;
	int numThreads = blockDim.x * gridDim.x;
	int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
	int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;
	float myElt = v[myIdx];
	float myLeftElt = v[myLeftIdx];
	float myRightElt = v[myRightIdx];
	v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}
//Faster kernel using shared memory 
//upto 2.4x faster on GEFORCE 920m
__global__ void smooth(float *arr,float *out)
{
	extern __shared__ float smem[];
	int tid = threadIdx.x;
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	int block = blockIdx.x;
	int mb = gridDim.x;
	smem[tid + 1] = arr[gid];
	if (block == 0)
	{
		if (tid == 0)
		{
			smem[0] = smem[1];
		}
	}
	else
	{
		if (tid == 0)
			smem[0] = arr[gid - 1];
	}
	if (block == mb-1){
		if (tid == N - 1)
		{
			smem[tid + 2] = arr[tid + 1];
		}
		else
		{
			smem[tid + 2] = arr[gid + 1];
		}
	}
	__syncthreads();
	out[gid] = smem[tid] * 0.25f + smem[tid+1] * 0.5f + smem[tid + 2] * 0.25f;
}

__global__ void warmup()
{
	//warmup kernel to launch gpu kernels quickly afterwards
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int x = id * 2;
	
}
int main()
{
	float *d_in, *d_out;
	float h_in[4096],h_out[4096];
	for (int i = 0; i < 4096; i++) {
		h_in[i] = (float)rand() / (float)RAND_MAX;;
	}
	hipMalloc((float**)&d_in, 4096 * sizeof(float));
	hipMemcpy(d_in, h_in, sizeof(float)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMalloc((float**)&d_out, 4096 * sizeof(float));
	warmup << <1, 1024 >> >();
	GpuTimer timer,timer2;
	timer.Start();
	smooth_naive << < 16, 256 >> >(d_in, d_out);
	timer.Stop();
	cout << timer.Elapsed() << endl;
	//hipError_t err = hipGetLastError();
	//cout << hipGetErrorString(err) << endl;
	timer2.Start();

	smooth <<< 16, 256, 258 * sizeof(float) >> >(d_in, d_out);
	timer2.Stop();
	cout << timer2.Elapsed() << endl;
	hipMemcpy(h_out, d_out, sizeof(float)*ARRAY_SIZE, hipMemcpyDeviceToHost);
	//for (int i = 0; i < 4096; i++)
	//	cout << h_out[i] << " ";

	hipFree(d_in);
	hipFree(d_out);
	hipDeviceReset();
	
	return 0;
}
