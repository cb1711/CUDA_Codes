#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
using namespace std;
__global__ void transpose(int *arr,int *out)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	int idy = blockDim.x*threadIdx.x + blockIdx.x;
	out[idy] = arr[idx];
}
//Transpose kernel using shared memory and optimized read and write pattern
//Performance much higher compared to the first transpose kernel
__global__ void transposeShared(int* arr, int *out)
{
	//Size of share[][] is 32X34 instead if 32X32 so as to avoid bank conflicts
	__shared__ int share[32][32+2];
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	int idy = blockDim.y*blockIdx.y + threadIdx.y;
	int index = idy*blockDim.x*gridDim.x + idx;
	int tix = threadIdx.x;
	int tiy = threadIdx.y;
	share[tiy][tix] = arr[index];
	__syncthreads();
	int in = tiy*blockDim.x + tix;
	int ir = in / blockDim.y;
	int ic = in % blockDim.y;
	int idyn = blockDim.x*blockIdx.x + ir;
	int idxn = blockDim.y*blockIdx.y + ic;
	//Writes to global memory are coalesced
	int tindex = idyn*blockDim.x*gridDim.x d+ idxn;
	out[tindex] = share[ic][ir];
}
//Kernel to generate the matrix 
__global__ void generate(int *arr)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	arr[idx] = idx;
}

int main()
{
	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	int *d_arr;
	hipMalloc((int**)&d_arr, 1024 * 1024 * sizeof(int));
	dim3 blocksize(1024);
	dim3 gridsize(1024);
	dim3 b2(32, 32);
	dim3 g2(32, 32);

	generate <<< gridsize, blocksize >>> (d_arr);
	int *d_out;
	hipMalloc((int**)&d_out, 1024 * 1024 * sizeof(int));
	//transpose << <gridsize, blocksize >> >(d_arr, d_out);
	transposeShared <<< g2, b2 >>>(d_arr, d_out);
	int *out;
	out = (int*)malloc(sizeof(int) * 1024 * 1024);
	hipMemcpy(out, d_out, sizeof(int) * 1024 * 1024, hipMemcpyDeviceToHost);
	for (int i = 0; i < 16; i++)
		for (int j = 0; j < 16; j++)
			cout << out[i*1024+j] << " ";
	hipDeviceReset();
	return 0;
}
